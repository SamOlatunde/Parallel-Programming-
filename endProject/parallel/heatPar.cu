#include "hip/hip_runtime.h"
//*****************************************************************
// End of Semester Project
// Name: Samuel Olatunde , and Sunil Rasaily
// GPU Programming Date: Date of Submission (11/28/2022)
//******************************************************************
// This solution uses global synchronization. The down side is that
// we are limited in the number execution resources available
// because we need all blocks/threads to be in the gpu at the time 
// of sync so as to avoid deadlock 
//******************************************************************
#include<iostream>
#include<hip/hip_runtime.h>
#include<cstdlib>
#include<hip/hip_runtime.h>
#include"timer.h"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
//error tolerance
const float eT  = 0.00001;

// data size
#define N  32

// limit for the max number of iterations
#define limit 100


#define blocks 1
#define threads 32

dim3 gridSize(blocks, blocks, 1);
dim3 blockSize(threads, threads, 1);


void print(float * a)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%f\t", a[i* N + j]);
        }

        printf("\n\n");
    }
}

double checkSum(float * h)
{
    double sum = 0.0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            sum += h[i*N+j];
        }
    }

    return sum;
}

__device__ int Continue = 0;

// Function Prototypes
__global__ void calcIntTempDistribution(float * h,float *g);
__device__ int converged (float newValue, float oldValue );
void initMetalPlate(float *h, float * g,float edgeTemp);

int main()
{
   // variable declarations
   float * h, *g ;

   //Allocate dynamic memory in host
   h = (float *) malloc ((N*N) * sizeof(float));
   g = (float *) malloc((N*N) * sizeof(float));

   float edgeTemp =70.5;//300;
   double tStart = 0.0, tStop = 0.0, tElapsed = 0.0;

   //initialize matrix
   initMetalPlate(h,g, edgeTemp);

   //device variables
   float *  hd,  * gd;
   long long int size = (N*N) * sizeof(float);

   // allocate space, and copy data
   hipMalloc((void**) & hd, size);
   hipMemcpy(hd, h, size, hipMemcpyHostToDevice);
   hipMalloc((void**) & gd, size);
   hipMemcpy(gd, g, size, hipMemcpyHostToDevice);

   //set grid dimensions
   /*dim3 dimGrid(8,8,1);
   dim3 dimBlock(32,32,1);*/

   //int THREADS;
   //int BLOCKS; 
   
   //hipOccupancyMaxPotentialBlockSize(&BLOCKS,&THREADS, calcIntTempDistribution, 0,0);
   //printf("Number of Blocks %d\n", BLOCKS);
   //printf("Number of Threads %d\n", THREADS);
  
   // kernel launch and timing
   GET_TIME(tStart);
   calcIntTempDistribution<<<gridSize, blockSize >>>(hd,gd);
   hipDeviceSynchronize();
   GET_TIME(tStop);

   hipMemcpy(h, hd, size, hipMemcpyDeviceToHost);

   // Compute how long it took
   tElapsed = tStop - tStart;

   printf("The code to be timed took %e seconds\n", tElapsed);
   printf("checkSum: %f\n", checkSum(h));
   //print(h);

   //free global memory
   hipFree(hd);
   hipFree(gd);

   // Dellocate dynamic memory
   free(h);
   free(g);

    return 0;
}



//*******************************************************************
// Name::calcIntTempDistribution()
// Parameters: 2 float pointers
//
//********************************************************************
__global__ void calcIntTempDistribution(float * h,float *g)
{
   int iteration = 0;
   int row = blockDim.y * blockIdx.y + threadIdx.y;
   int col = blockDim.x * blockIdx.x + threadIdx.x;

   //number of threads in grid
   int totalThreadsX = blockDim.x * gridDim.x;
   int totalThreadsY = blockDim.y * gridDim.y;
   
   //auto grid = cg::this_grid();

   do
   {
        for (int i = row; i < N; i += totalThreadsY)
        {
            for (int j = col; j < N; j += totalThreadsX)
            {
                //Takes care of boundary points
                if (i != 0 && i != (N - 1) && j != 0 && j != (N - 1))
                {
                    g[i * N + j] = 0.25 * (h[(i - 1) * N + j] + h[(i + 1) * N + j] +
                                            h[i * N + j - 1] + h[i * N + j + 1]);
                }

            }
        }
        
        //synchronize the whole grid 
       // grid.sync();

        Continue = 0;

        for (int i = row; i < N; i += totalThreadsY)
        {
            for (int j = col; j < N; j += totalThreadsX)
            {
                if (converged(g[i * N + j], h[i * N + j]) == 0)
                {
                    Continue = 1;
                }

                h[i * N + j] = g[i * N + j];
            }
        }
        
        //sync grid again to avoid race condition
        // grid.sync();

     iteration++;
   }while(Continue == 1 && iteration < limit);
   //printf("Blah");
   //printf("%d\n", iteration);
}

//*******************************************************************
// Name::converged()
// Parameters: 2 floats
// Tests for convergence of two points. Returns true if the error is
// within error tolerance; false otherwise
//********************************************************************
// bool converged (float newValue, float oldValue )
 __device__ int converged (float newValue, float oldValue )
{
    float er = (newValue-oldValue)/newValue;
    //printf("er %f\n", er);
    if (er < 0) er = -er;

    if (er <= eT)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}


//*******************************************************************
// Name::initMetalPlate()
// Parameters: 1 2d float array, 1 float
// Initializes the metal sheet with the intial values of the edges
// and guess values for interior points
//********************************************************************
void initMetalPlate(float *h, float * g,float edgeTemp)
{
   //we reduce the temparture by this value with every
   // outer loop iteration
   float reduceFactor = edgeTemp/N;

   int row = 0;
   int col;

   for( int i = 0; i < (N/2); i++)
   {

        row = i;
        for (col = i; col < N-i; col++ )
        {
            h[row * N + col] = edgeTemp;
            g[row * N + col] = edgeTemp;
        }

        col--;

        for (row = row+1; row < N-i; row++)
        {
            h[row * N + col] = edgeTemp;
            g[row * N + col] = edgeTemp;
        }

        row--;
        col--;

        for(col = col; col >=i; col--)
        {
           h[row * N + col] = edgeTemp;
           g[row * N + col] = edgeTemp;
        }

        row--;
        col++;

        for(row = row; row >i; row--)
        {
            h[row * N + col] = edgeTemp;
            g[row * N + col] = edgeTemp;
        }

      edgeTemp = edgeTemp - reduceFactor;
    }

    // print(g);
    // printf("\n\n");

}